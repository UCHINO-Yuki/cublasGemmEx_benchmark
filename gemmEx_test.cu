#include <iostream>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <fstream>
#include <cxxabi.h>
#include <chrono>

hipEvent_t start, stop;

void startTimer() {
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
}

float stopTimer() {
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return milliseconds;
}

template <typename T>
__global__ void set_mat_kernel(std::size_t m, std::size_t n, T *Mat, T value) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= m*n) {
		return;
	}
	Mat[tid] = value;
}

void test_sq_mat(int rep, std::size_t nmin, std::size_t nmax){
    // setting for cublasGemmEx
    // Change yourself!!
    using IN = std::int8_t;
    using OUT = std::int32_t;
    const auto typeIN = HIP_R_8I;
    const auto typeOUT = HIP_R_32I;
    const auto CompMode = HIPBLAS_COMPUTE_32I;

    // handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // set scalars
    const OUT alpha = 1;
    const OUT beta = 0;

    // set matrices
    std::size_t sizeAmax = nmax * nmax;
    IN *d_A, *d_B;
    OUT *d_C;
    hipMalloc(&d_A, sizeAmax * sizeof(IN));
    hipMalloc(&d_B, sizeAmax * sizeof(IN));
    hipMalloc(&d_C, sizeAmax * sizeof(OUT));
	dim3 threads = 256;	// <= 1024
	dim3 grid = (sizeAmax + 256 - 1) / 256;
	set_mat_kernel<IN> <<< grid, threads >>> (nmax, nmax, d_A, 1);
	set_mat_kernel<IN> <<< grid, threads >>> (nmax, nmax, d_B, 1);
	set_mat_kernel<OUT> <<< grid, threads >>> (nmax, nmax, d_C, 0);

    // evaluate performance
    for(std::size_t n = nmin; n <= nmax; n <<= 1){
        std::ofstream file("gemmEx_test_result.csv", std::ios::app);

        // warm up
        hipblasStatus_t stat;
        for (int j = 0; j < 10; j++) {
            stat = hipblasGemmEx(handle,
                                HIPBLAS_OP_T, HIPBLAS_OP_N,
                                n, n, n,
                                &alpha,
                                d_A, typeIN, n,
                                d_B, typeIN, n,
                                &beta,
                                d_C, typeOUT, n,
                                CompMode, HIPBLAS_GEMM_DEFAULT);
        }
        if(stat != HIPBLAS_STATUS_SUCCESS){
            std::cout << "hipblasGemmEx failed" << std::endl;
        }

        hipError_t err = hipPeekAtLastError();
        if (err != hipSuccess) {
            std::printf("Kernel launch error: %s\n", hipGetErrorString(err));
            return ;
        }
        
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            std::printf("Kernel execution error: %s\n", hipGetErrorString(err));
            return ;
        }
        
        float ms = 0.0;
        for(int j = 0; j < rep; j++){
            startTimer();

            stat = hipblasGemmEx(handle,
                HIPBLAS_OP_T, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                d_A, typeIN, n,
                d_B, typeIN, n,
                &beta,
                d_C, typeOUT, n,
                CompMode, HIPBLAS_GEMM_DEFAULT);
            
            ms += stopTimer();
             
            if(stat != HIPBLAS_STATUS_SUCCESS){
                std::cout << "hipblasGemmEx failed" << std::endl;
            }

            hipError_t err = hipPeekAtLastError();
            if (err != hipSuccess) {
                std::printf("Kernel launch error: %s\n", hipGetErrorString(err));
                return ;
            }
     
            err = hipDeviceSynchronize();
            if (err != hipSuccess) {
                std::printf("Kernel execution error: %s\n", hipGetErrorString(err));
                return ;
            }
        }
        hipDeviceSynchronize();

        ms /= rep;
        int status;
        std::printf("hipblasGemmEx input: %s, output: %s, %d x %d x %d takes %e ms, TOPS is %e\n", 
            abi::__cxa_demangle(typeid(IN).name(),0,0,&status),
            abi::__cxa_demangle(typeid(OUT).name(),0,0,&status),
            (int)n, (int)n, (int)n, 
            ms, 2.0*n*n*n/ms/1e9);

        file << n << "," << ms << "\n";
        file.close();
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
    hipDeviceReset();
}

int main(int argc, char *argv[]){
    // # of iteration
    int rep = 50;
    if (argc > 1) {
        rep = atoi(argv[1]);
    }
    
    // min of log2(n)
    std::size_t nmin = (std::size_t)std::pow(2,8);
    if (argc > 2) {
        nmin = (std::size_t)std::pow(2,atoi(argv[2]));
    }
    
    // max of log2(n)
    std::size_t nmax = (std::size_t)std::pow(2,16);
    if (argc > 3) {
        nmax = (std::size_t)std::pow(2,atoi(argv[3]));
    }

    // execution
    test_sq_mat(rep,nmin,nmax);

    return 0;
}